#include <object.cuh>
#include <spheres.cuh>
using namespace std;

ObjectsGPUAdapter::ObjectsGPUAdapter(
    const vector<Object*> &objects) {

  // Init GPU point & assosiated pointers

  int nObjs = objects.size();
  DSphere **htod_objs, **sphs;

  hipMalloc((void **)&sphs, nObjs*sizeof(DSphere*));
  htod_objs = (DSphere**)malloc(nObjs*sizeof(DSphere*));

  for (auto &obj: objects)
    obj_adapters.push_back(new SphereGPUAdapter((Sphere*)obj));

  for (int i=0; i<nObjs; i++)
    htod_objs[i] = obj_adapters[i]->getDS();

  // Copy host pointers to GPU

  hipMemcpy(sphs, htod_objs, nObjs*sizeof(DSphere*),
      hipMemcpyHostToDevice);

  h_sphs = (DSpheres*)malloc(sizeof(DSpheres));
  hipMalloc((void **)&d_sphs, sizeof(DSpheres));
  h_sphs->s = sphs;
  h_sphs->nSpheres = nObjs;

  hipMemcpy(d_sphs, h_sphs, sizeof(DSpheres),
      hipMemcpyHostToDevice);

  free(htod_objs);
}

ObjectsGPUAdapter::~ObjectsGPUAdapter(void) {

  for (int i=0; i<obj_adapters.size(); i++)
    delete obj_adapters[i];

  hipFree(d_sphs); hipFree(h_sphs->s); free(h_sphs);
}
